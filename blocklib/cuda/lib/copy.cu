#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>



__global__ void
apply_copy_kernel(hipFloatComplex* in, hipFloatComplex* out, int batch_size, int load = 1)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int n = batch_size;
    if (i < n) {
        for (int x = 0; x < load; x++) {
            out[i].x = in[i].x;
            out[i].y = in[i].y;
        }
    }
}

void apply_copy(hipFloatComplex* in,
                hipFloatComplex* out,
                int grid_size,
                int block_size,
                int load,
                hipStream_t stream)
{
    int batch_size = block_size * grid_size;
    apply_copy_kernel<<<grid_size, block_size, 0, stream>>>(in, out, batch_size, load);
}

void get_block_and_grid(int* minGrid, int* minBlock)
{
    hipOccupancyMaxPotentialBlockSize(minGrid, minBlock, apply_copy_kernel, 0, 0);
}